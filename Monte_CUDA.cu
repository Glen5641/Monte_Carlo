#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>

/*
 * Monte Carlo Pi Estimation Algorithm in CUDA
 *
 * This Project uses Cuda and thread
 * topology to estimate Pi.
 *
 * Author: Clayton Glenn
 */

#define MAX_THREAD 16
#define MIN_THREAD 8
#define MAX_N 20
#define MIN_N 8
#define BLOCK_SIZE 256
#define DEBUG 0

/** Kernel Function
  * First finds the Thread ID within the block of GPU Threads
  * and if the Thread is Correct, it Encrypts the corresponding
  * Character in the String.
 **/
__global__
void monte(int *flags, float *x_vals, float *y_vals, int t, int n) {

  //Get Thread id
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  // Loop N/Threads times plus one
  for(int i = 0; i < (n/t + 1); i++){

    // If looped id count is less than n, grab rand x
    // and y and check within unit. Increment if so
    if((i*t+tid) < n){
      if((pow(x_vals[(i*t+tid)], 2) + pow(y_vals[(i*t+tid)],2)) <= 1) flags[(tid)]++;
    }
  }
}

/**
  * Helper Function
  * Prints an string to standard error showing help
  * for valid arguments in the executable
 **/
void printerror(){
  fprintf(stderr, "Invalid Arguments\n");
  fprintf(stderr, "Correct Form: ./monte [# threads] [# points]\n");
  exit(0);
}
/**
  * Main Program
  * This Program is for Homework 6 to encrypt some text or show
  * the encryption method of text that is 2 to the power of N
  * characters long all initialized to zero.
 **/
int main(int argc, char **argv) {

  // Declare a buffer of max size to start
  int N       = MIN_THREAD;
  int THREADS = MIN_THREAD;
  int BLOCKS  = 256;

  // Check for immediate errors in args
  if (argc < 3 || argc > 3) printerror();

  // Get Thread Count Per Block
  THREADS = strtol(argv[1], NULL, 10);
  THREADS = ((int)pow(2, THREADS));
  if(THREADS < BLOCKS) BLOCKS = 1;
  else THREADS = THREADS / BLOCKS;

  // Get N Coordinates
  N = strtol(argv[2], NULL, 10);
  N = (int)pow(2, N);

  // Print N and Threads for distinguish
  printf("(Threads: %d) (N: %d)\n", THREADS * BLOCKS, N);

  //Set Array of Size Thread
  int flags[BLOCKS*THREADS];
  float randx[N];
  float randy[N];
  srand( time( NULL ) );
  for(int i = 0; i < N; i++){
    if(i < BLOCKS*THREADS)flags[i] = 0;
    randx[i] = ( float )rand()/RAND_MAX;
    randy[i] = ( float )rand()/RAND_MAX;
  }

  // Init all other variables
  int *dev_flags;
  float *dev_randx;
  float *dev_randy;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float final_time = 0.0;

  // Allocate memory in the GPU for the int array
  hipMalloc(&dev_randx, N*sizeof(float));
  hipMalloc(&dev_randy, N*sizeof(float));
  hipMalloc(&dev_flags, BLOCKS*THREADS*sizeof(int));

  // Copy the Memory from the array to the array pointers
  hipMemcpy(dev_flags, flags, BLOCKS*THREADS*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_randx, randx, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_randy, randy, N*sizeof(float), hipMemcpyHostToDevice);

  // Total Time Record
  hipEventRecord(start);
  monte<<<BLOCKS, THREADS>>>(dev_flags, dev_randx, dev_randy, BLOCKS*THREADS, N);
  hipEventRecord(stop);

  // Copy the results from GPU to the CPU
  hipMemcpy(flags, dev_flags, BLOCKS*THREADS*sizeof(int), hipMemcpyDeviceToHost);

  // Count total successes for each thread
  int success = 0;
  for(int i = 0; i < BLOCKS*THREADS; i++){
    if(flags[i] > 0) success += flags[i];
  }

  // Print Successes, failures, and estimation
  //printf("Success: %d\n", success);
  //printf("Failure: %d\n", (N - success));
  printf("Estimation of Pi: %1.6f\n", ((float)success/N)*4);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&final_time, start, stop);
  printf("Time in Kernel: %1.10f\n\n", final_time/1000);

  hipFree(dev_flags);
  hipFree(dev_randx);
  hipFree(dev_randy);
}
